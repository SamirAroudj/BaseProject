/**
	Author: Samir Aroudj
*/
#include <cassert>
#include "FailureHandling.h"

void FailureHandling::checkForCUDAErrors()
{	
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
	{
		const char *errorString = hipGetErrorString(hipError_t);
		assert(false);
		if (errorString)
			errorString = NULL;
	}
}
