
#include <hip/hip_runtime.h>
/**
	Author: Samir Aroudj
*/

#ifdef CUDA

#include <cstring>
#include "Platform/Profiling/CUDAProfiler.h"
#include "Platform/FailureHandling/FailureHandling.h"

using namespace Profiling;
using namespace FailureHandling;
using namespace std;

CUDAProfiler::EventPair::EventPair()
{
	memset(this, 0, sizeof(EventPair));

	cudaEventCreateWithFlags(&mStart, cudaEventDefault);
	cudaEventCreateWithFlags(&mEnd, cudaEventDefault);
}

CUDAProfiler::CUDAProfiler(const string *measurementNames, uint32 numberOfMeasurementNames) :
	Profiler(measurementNames, numberOfMeasurementNames)
{
	#ifdef PROFILING
		// create an event pair for each measurement type
		for (uint32 i = 0; i < numberOfMeasurementNames; ++i)
			mEventPairs.push_back(EventPair());
	#endif // PROFILING
}

CUDAProfiler::~CUDAProfiler()
{
	#ifdef PROFILING
		uint32 pairCount = (uint32) mEventPairs.size();

		for (uint32 i = 0; i < pairCount; ++i)
		{
			EventPair &pair = mEventPairs[i];

			cudaEventDestroy(pair.mStart);
			cudaEventDestroy(pair.mEnd);
		}
	#endif // PROFILING
}

uint32 CUDAProfiler::addMeasurementType(const string &name)
{
	#ifdef PROFILING
		// create the event pair
		mEventPairs.push_back(EventPair());

		return Profiler::addMeasurementType(name);

	#else
		return 0;

	#endif // PROFILING
}

void CUDAProfiler::recordEndEvent(uint32 index, const cudaStream_t &stream)
{
	#ifdef PROFILING
		assert(index < mEventPairs.size());

		EventPair &pair = mEventPairs[index];
		cudaEventRecord(pair.mEnd, stream);
		pair.mSetEnd = true;
	#endif // PROFILING
}

void CUDAProfiler::recordStartEvent(uint32 index, const cudaStream_t &stream)
{
	#ifdef PROFILING
		assert(index < mEventPairs.size());

		EventPair &pair = mEventPairs[index];
		cudaEventRecord(pair.mStart, stream);
		pair.mSetStart = true;
	#endif // PROFILING
}

void CUDAProfiler::measureEventTiming(uint32 index)
{
	#ifdef PROFILING
		assert(index < mEventPairs.size());
		float		milliseconds = 0;
		EventPair	&pair		 = mEventPairs[index];

		// Did the user record both events?
		assert(pair.mSetStart);
		assert(pair.mSetEnd);

		// compute delta time
		cudaEventElapsedTime(&milliseconds, pair.mStart, pair.mEnd);
		pair.mSetEnd = false;
		checkForErrors();

		// update statistics
		double seconds = milliseconds / 1000.0;
		addTimeMeasurement(index, seconds);
	#endif // PROFILING
}

void CUDAProfiler::measureEventTimings()
{
	#ifdef PROFILING
		// get measurement for each recorded pair
		uint32 pairCount = (uint32) mEventPairs.size();
		for (uint32 pairIdx = 0; pairIdx < pairCount; ++pairIdx)
		{
			// get recorded pair
			EventPair &pair = mEventPairs[pairIdx];
			if (!pair.mSetEnd || !pair.mSetStart)
				continue;

			measureEventTiming(pairIdx);
		}
	#endif // PROFILING
}

#endif // CUDA
